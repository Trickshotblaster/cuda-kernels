#include "hip/hip_runtime.h"
// simple cuda kernel to add together vectors a and b, storing the result in c
// to run:
// nvcc add.cu && ./a.out

// for printing
#include <stdio.h>

// addition kernel
__global__ void add_vectors(float (*a)[], float (*b)[], float (*c)[], int n) {
    // get the current thread index
    // the kernel is run in parallel on a bunch of threads
    // from the perspective of a single thread
    int id = threadIdx.x;
    // basic sanity checks, make sure kernel is being called and values are correct
    printf("\nHello World from thread %d", id);
    printf("\nValue of a at thread: %f", (*a)[id]);

    // what's happening:
    // each thread adds one value
    // instead of [0.1, 0.2, 0.3, 0.4, 0.5] + [0.1, 0.2, 0.3, 0.4, 0.5] -> result [...],
    // thread 0 sees 0.1 + 0.1 = 0.2 -> c[0] = 0.2
    // thread 1 sees 0.2 + 0.2 = 0.4 -> c[1] = 0.4 and so on

    // check that the current thread index is within the length of the vectors
    if (id < n) {
        // if it is, write a[id] + b[id] into c[id]
        (*c)[id] = (*a)[id] + (*b)[id];
    }
    // log the output for debugging
    printf("\nOutput for thread: %f", (*c)[id]);

    // note that nothing is retuned, it is simply written to memory and later copied
}

int main() {
    printf("Program started");
    // define length of vectors
    // must be a const because compiler doesn't like variable length arrays
    const int n = 5;
    // size of vectors in memory (bytes)
    const int size = n * sizeof(float);
    // make some random vectors to test with, r_c is the output
    float r_a[n] = {0.1, 0.2, 0.3, 0.4, 0.5};
    float r_b[n] = {0.1, 0.2, 0.3, 0.4, 0.5};
    float r_c[n] = {0.0};
    // https://www.codeproject.com/Articles/7042/How-to-interpret-complex-C-C-declarations
    // make some empty pointer variables before malloc-ing on host and device, assigning, and memcpying
    float (*h_a)[n];
    float (*d_a)[n];

    float (*h_b)[n];
    float (*d_b)[n];

    float (*h_c)[n];
    float (*d_c)[n];
    // verify the size of memory we are allocating
    printf("\nSize: %d", size);
    printf("\nSize of r_a (bytes): %lu", sizeof(r_a));

    // https://stevengong.co/notes/CUDA-Memory-Allocation
    // allocate on device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // allocate on host
    hipHostMalloc(&h_a, size);
    hipHostMalloc(&h_b, size);
    hipHostMalloc(&h_c, size);
    // assign variables (just pointers, will be read inside of kernel)
    h_a = &r_a;
    h_b = &r_b;
    h_c = &r_c;
    // make sure nothing has gone wrong after cuda malloc on the host
    printf("\nFirst element of r_a after cuda malloc host: %f", r_a[0]);
    printf("\nFirst element of h_a after cuda malloc host: %f", *h_a[0]);
    // copy h_a, h_b, and h_c to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);
    // make sure everything made it there safely
    printf("\nFirst element of r_a after cuda memcpy: %f", r_a[0]);
    printf("\nFirst element of h_a after cuda memcpy: %f", *h_a[0]);
    // call the actual kernel
    // done with an extra thread to check if going out of boundaries works
    add_vectors<<<1,n+1>>>(d_a, d_b, d_c, n);
    // copy arrays back to the host so they can be read and printed from main()
    hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // print all items of c and make sure they are correct
    printf("\nAll entries of c:\n");
    for (int i=0; i<n; i++) {
        printf("%f", (*h_c)[i]);
        if (abs((r_a[i] + r_b[i]) - (*h_c)[i]) < 1e-4) {
            printf(" -- Correct");
        } else {
            printf(" -- Incorrect, should have been %f", r_a[i] + r_b[i]);
        }
        printf("\n");
    }

    // free memory (don't use regular free here or it will give invalid address)
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}